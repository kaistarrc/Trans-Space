#include "hip/hip_runtime.h"
#pragma once
//cv
#include <opencv2/core.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <opencv2/opencv.hpp>

#pragma comment(lib,"opencv_world310.lib")

//gl
#include "GL\glew.h"
#include "GL\freeglut.h"

//cuda
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>
#include "hip/hip_runtime_api.h"
#include <hip/hip_runtime.h>
#include <>
#include <hip/device_functions.h>
#include <malloc.h>
#include <hipblas.h>
#include <npp.h>

//gl
#pragma comment(lib,"glew32.lib")
#pragma comment(lib,"glew32s.lib")

//cv
#pragma comment(lib,"opencv_world310.lib")

//cuda
#pragma comment(lib,"cuda.lib")
#pragma comment(lib,"cudart.lib")
#pragma comment(lib,"cublas.lib")

#pragma comment(lib,"nppi.lib")
#pragma comment(lib,"npps.lib")
#pragma comment(lib,"nppc.lib")

texture<uchar4, 2, hipReadModeElementType> color_tex;
texture<float4, 2, hipReadModeElementType> extra_tex;

static int width;// = 128;
static int height;// = 128;
static int block_numx;// = 4;
static int block_numy;// = 4;
static int particle_numx;// = 8;
static int particle_numy;// = 4;

static float* cost_dif_cu;
static float* cost_dif_reduce_cu;

static float* cost_and_cu;
static float* cost_and_reduce_cu;

static float* cost_or_cu;
static float* cost_or_reduce_cu;


static float* img_dif_cu;
static float* img_and_cu;
static float* img_or_cu;



extern "C" void kernel_bind_CUDA(hipArray_t array1, hipArray_t array2)
{
	hipBindTextureToArray(color_tex, array1, color_tex.channelDesc);
	hipBindTextureToArray(extra_tex, array2, extra_tex.channelDesc);
}

extern "C" void kernel_unbind_CUDA()
{
	hipUnbindTexture(color_tex);
	hipUnbindTexture(extra_tex);
}

extern "C" void initCudaMem(int w,int h,int p_numx,int p_numy,int b_numx,int b_numy)
{
	width = w;	height = h;
	particle_numx = p_numx;	particle_numy = p_numy;
	block_numx = b_numx;	block_numy = b_numy;
	
	
	hipMalloc(&img_dif_cu, sizeof(float)*w*p_numx*h*p_numy);
	hipMalloc(&img_and_cu, sizeof(float)*w*p_numx*h*p_numy);
	hipMalloc(&img_or_cu, sizeof(float)*w*p_numx*h*p_numy);

	hipMalloc(&cost_dif_reduce_cu, block_numx*block_numy*particle_numx*particle_numy * sizeof(float));
	hipMalloc(&cost_and_reduce_cu, block_numx*block_numy*particle_numx*particle_numy * sizeof(float));
	hipMalloc(&cost_or_reduce_cu, block_numx*block_numy*particle_numx*particle_numy * sizeof(float));

	hipMalloc(&cost_dif_cu, sizeof(float)*particle_numx*particle_numy);
	hipMalloc(&cost_and_cu, sizeof(float)*particle_numx*particle_numy);
	hipMalloc(&cost_or_cu, sizeof(float)*particle_numx*particle_numy);
	
}

extern "C" void releaseCudaMem()
{
	hipFree(cost_dif_reduce_cu);

}

extern "C" void getReduceResult(float* cost_dif_reduce,float* cost_and_reduce,float* cost_or_reduce)
{
	hipMemcpy(cost_dif_reduce, cost_dif_reduce_cu, sizeof(float)* 4 * particle_numx * 4 * particle_numy, hipMemcpyDeviceToHost);
	hipMemcpy(cost_and_reduce, cost_and_reduce_cu, sizeof(float)* 4 * particle_numx * 4 * particle_numy, hipMemcpyDeviceToHost);
	hipMemcpy(cost_or_reduce, cost_or_reduce_cu, sizeof(float)* 4 * particle_numx * 4 * particle_numy, hipMemcpyDeviceToHost);

}
extern "C" void getCostFromGPU(cv::Mat& cost_dif,cv::Mat& cost_and,cv::Mat& cost_or)
{
	hipMemcpy(cost_dif.data, cost_dif_cu, sizeof(float)*particle_numx*particle_numy, hipMemcpyDeviceToHost);
	hipMemcpy(cost_and.data, cost_and_cu, sizeof(float)*particle_numx*particle_numy, hipMemcpyDeviceToHost);
	hipMemcpy(cost_or.data, cost_or_cu, sizeof(float)*particle_numx*particle_numy, hipMemcpyDeviceToHost);

}
extern "C" void getDifferenceImageFromGPU(cv::Mat& dif,cv::Mat& and,cv::Mat& or)
{
	int s = width*particle_numx*height*particle_numy;

	hipMemcpy(dif.data, img_dif_cu, sizeof(float)*s, hipMemcpyDeviceToHost);
	hipMemcpy(and.data, img_and_cu, sizeof(float)*s, hipMemcpyDeviceToHost);
	hipMemcpy(or.data, img_or_cu, sizeof(float)*s, hipMemcpyDeviceToHost);

}

__global__ void differentiate(float* o, float* dif,float* and,float* or, int _w, int _h, int _pnumx, int _pnumy,float dif_max)
{
	unsigned int width_full = _w*_pnumx;  //width*particle_numx;
	unsigned int height_full = _h*_pnumy; //height*particle_numy;

	//index in an image
	unsigned int tx = threadIdx.x;
	unsigned int ty = threadIdx.y;
	unsigned int bw = blockDim.x;
	unsigned int bh = blockDim.y;

	//index of a particle
	unsigned int px = blockIdx.z % _pnumx; //particle_numx;
	unsigned int py = blockIdx.z / _pnumx; //particle_numx;

	//global index in whole images.
	unsigned int u = px*_w + (bw*blockIdx.x) + tx;
	unsigned int v = py*_h + (bh*blockIdx.y) + ty;

	//working
	/*
	if (o[v*width_full + u] == 0)
		dif[v*width_full + u] = 0;
	else
	{
		dif[v*width_full + u] = abs(o[v*width_full + u] - tex2D(extra_tex, u, height_full - v-1).z);
		if (dif[v*width_full + u] > 100)
			dif[v*width_full + u] = 100;
	}
	*/
	
	//modified ( working in PSO ) 
	/*
	and[v*width_full + u] = 0;	or[v*width_full + u] = 0;	dif[v*width_full + u] = 0;

	if (o[v*width_full + u] > 0){
		dif[v*width_full + u] = abs(o[v*width_full + u] - tex2D(extra_tex, u, height_full - v - 1).z);

		if (dif[v*width_full + u] > dif_max)
			dif[v*width_full + u] = dif_max;
	}

	if (o[v*width_full + u] > 0 && tex2D(extra_tex, u, height_full - v - 1).z > 0)
	{
		if (dif[v*width_full + u] < dif_max)
			and[v*width_full + u] = 1;
	}

	if (o[v*width_full + u] > 0 || tex2D(extra_tex, u, height_full - v - 1).z > 0)
		or[v*width_full + u] = 1;
	*/

	//modifying (in HMF)

	and[v*width_full + u] = 0;	or[v*width_full + u] = 0;	dif[v*width_full + u] = 0;
	
	if (o[v*width_full + u] > 0 && tex2D(extra_tex, u, height_full - v - 1).z > 0){
	

		dif[v*width_full + u] = abs(o[v*width_full + u] - tex2D(extra_tex, u, height_full - v - 1).z);
		if (dif[v*width_full + u] > dif_max)
			dif[v*width_full + u] = dif_max;

		//if (dif[v*width_full + u] <dif_max)  // cost may be bigger than 1 due to this.
			and[v*width_full + u] = 1;
	}

	if (o[v*width_full + u] > 0 || tex2D(extra_tex, u, height_full - v - 1).z > 0)
		or[v*width_full + u] = 1;
	
	
}

__global__ void sum_and_reduce(float* g_diff, float* g_odata,int _w,int _h,int _pnumx,int _pnumy)
{
	extern __shared__ float sData[];

	unsigned int width_full = _w*_pnumx;  //width*particle_numx;
	unsigned int height_full = _h*_pnumy; //height*particle_numy;

	//index in an image
	unsigned int tx = threadIdx.x;
	unsigned int ty = threadIdx.y;
	unsigned int bw = blockDim.x;//=32
	unsigned int bh = blockDim.y;//=32

	//index of a particle
	unsigned int px = blockIdx.z % _pnumx; 
	unsigned int py = blockIdx.z / _pnumx; 

	//global index in whole images.
	unsigned int u = px*_w + (bw*blockIdx.x) + tx;
	unsigned int v = py*_h + (bh*blockIdx.y) + ty;

	//thread ID / global block ID 
	unsigned int tid = tx + ty*bw;
	unsigned int bidx = blockIdx.x + px *  gridDim.x;
	unsigned int bidy = blockIdx.y + py *  gridDim.y;
	unsigned int bid = bidx + bidy *  gridDim.x*_pnumx;


	sData[tid] = g_diff[v*width_full + u];
	__syncthreads();

	for (int s = 1; s < blockDim.x * blockDim.y; s *= 2)
	{
		if (tid % (2 * s) == 0)
		{
			//sData[tid] = (sData[tid] & 0x0000ff) + (sData[tid + s] & 0x0000ff);
			sData[tid] += sData[tid + s];

		}
		__syncthreads();
	}
	if (tid == 0) g_odata[bid] = sData[0];
}

__global__ void sum_and_reduce2(float *g_idata, float *g_odata) {
	extern __shared__ float sData[];

	//index in an image
	unsigned int tx = threadIdx.x;
	unsigned int ty = threadIdx.y;
	unsigned int bw = blockDim.x;//=4
	unsigned int bh = blockDim.y;//=4

	//index of a particle
	unsigned int px = blockIdx.x;
	unsigned int py = blockIdx.y;

	//global index in whole images.
	unsigned int u = px*bw + tx;
	unsigned int v = py*bh + ty;

	unsigned int width_full = bw*gridDim.x;
	unsigned int height_full = bh*gridDim.y;

	//thread ID / global block ID 
	unsigned int tid = tx + ty*bw;
	unsigned int bid = px + py*gridDim.x;

	sData[tid] = g_idata[v*width_full + u];
	__syncthreads();

	// do reduction in shared mem
	for (int s = 1; s < blockDim.x*blockDim.y; s *= 2) {
		if (tid % (2 * s) == 0) {
			sData[tid] += sData[tid + s];
		}
		__syncthreads();
	}
	// write result for this block to global mem
	if (tid == 0) g_odata[bid] =  sData[0];
	
}


extern "C" void calculatecost_cu(float* img_ob_cu,float dif_max)//, float* img_dif_cu)//,float* cost_cu)//,int g)
{
	dim3 block(width/block_numx, height/block_numy);  //=(32,32)
	dim3 grid(block_numx,block_numy, particle_numx*particle_numy);   //=(4,4,particle_num)

	// difference map	
	differentiate << <grid, block >> > (img_ob_cu, img_dif_cu,img_and_cu,img_or_cu,width,height,particle_numx,particle_numy,dif_max);
	hipDeviceSynchronize();

	// reduce
	int sbytes1 = block.x*block.y*sizeof(float);
	sum_and_reduce << <grid, block, sbytes1 >> > (img_dif_cu, cost_dif_reduce_cu,width,height,particle_numx,particle_numy);
	//hipDeviceSynchronize();
	
	sum_and_reduce << <grid, block, sbytes1 >> > (img_and_cu, cost_and_reduce_cu, width, height, particle_numx, particle_numy);
	//hipDeviceSynchronize();

	sum_and_reduce << <grid, block, sbytes1 >> > (img_or_cu, cost_or_reduce_cu, width, height, particle_numx, particle_numy);
	hipDeviceSynchronize();
	
	
	dim3 block2(block_numx,block_numy);  //=(4,4)
	dim3 grid2(particle_numx, particle_numy, 1);   //=(4,4,particle_num)

	int sbytes2 = block2.x*block2.y*sizeof(float);
	sum_and_reduce2 << <grid2, block2, sbytes2 >> > (cost_dif_reduce_cu, cost_dif_cu);
	//hipDeviceSynchronize();

	sum_and_reduce2 << <grid2, block2, sbytes2 >> > (cost_and_reduce_cu, cost_and_cu);
	//hipDeviceSynchronize();

	sum_and_reduce2 << <grid2, block2, sbytes2 >> > (cost_or_reduce_cu, cost_or_cu);
	hipDeviceSynchronize();

}

__global__ void get_texture_depth(float* dstBuffer, int _w, int _h, int _pnumx, int _pnumy)
{

	unsigned int width_full = _w*_pnumx;  //width*particle_numx;
	unsigned int height_full = _h*_pnumy; //height*particle_numy;

	unsigned int tx = threadIdx.x;
	unsigned int ty = threadIdx.y;
	unsigned int bw = blockDim.x;
	unsigned int bh = blockDim.y;

	unsigned int px = blockIdx.z % _pnumx; //particle_numx;
	unsigned int py = blockIdx.z / _pnumx; //particle_numx;

	unsigned int u = px*_w + (bw*blockIdx.x) + tx;
	unsigned int v = py*_h + (bh*blockIdx.y) + ty;

	dstBuffer[v*_w*_pnumx + u] = tex2D(extra_tex, u, height_full - v - 1).z;

}


extern "C" void get_depth_cu(float* img_cu)
{
	dim3 block(width / block_numx, height / block_numy);  //=(32,32)
	dim3 grid(block_numx, block_numy, particle_numx*particle_numy);   //=(4,4,particle_num)

	get_texture_depth << <grid, block >> >(img_cu, width, height, particle_numx, particle_numy);
	hipDeviceSynchronize();
}


